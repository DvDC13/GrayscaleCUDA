#include "hip/hip_runtime.h"
#include "grayscaleGPU.cuh"
#include "pixel.h"

#define channels 3

__global__ void ConvertToGrayscaleGPU(unsigned char* imageRGBA)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int gridIdx = x + y * blockDim.x * gridDim.x;

    Pixel* pixel = reinterpret_cast<Pixel*>(&imageRGBA[gridIdx * channels]);
    unsigned char pixelGray = static_cast<unsigned char>(0.2126f * pixel->red + 0.7152f * pixel->green + 0.0722f * pixel->blue);
    pixel->red = pixelGray;
    pixel->green = pixelGray;
    pixel->blue = pixelGray;
}